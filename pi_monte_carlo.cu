#include <iostream>
#include <vector>
#include <iomanip>
#include <hip/hip_runtime.h> // for cuda functions
#include <time.h> // for time(NULL)
#include <cmath> // for std::abs and std::acos
#include <stdexcept> // for std::invalid_argument
#include <string> // for std::stoull
#include <hiprand/hiprand_kernel.h> // for curand functions

#define CUDA_CHECK(err) { \
    hipError_t err_ = (err); \
    if (err_ != hipSuccess) { \
        fprintf(stderr, "CUDA error at %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err_)); \
        exit(EXIT_FAILURE); \
    } \
}

#define THREADS_PER_BLOCK 256

__global__ void estimate_pi_kernel(unsigned long long seed, unsigned long long samples_per_thread, unsigned long long* block_results) {
    // shared memory for reduction within a block
    __shared__ unsigned long long shared_counts[THREADS_PER_BLOCK];

    // get thread and block indices
    unsigned int tid = threadIdx.x;
    unsigned int block_id = blockIdx.x;
    unsigned int global_thread_id = block_id * blockDim.x + tid;

    // Use unsigned long long for the local count to prevent overflow if samples_per_thread
    // exceeds the limit of unsigned int (~4.29 billion).
    unsigned long long local_count = 0;

    // initialize random number generator state for each thread
    hiprandState_t state;
    hiprand_init(seed + global_thread_id, 0, 0, &state);

    // each thread generates its assigned number of random points
    for (unsigned long long i = 0; i < samples_per_thread; ++i) {
        // generate random x and y coordinates between 0.0 and 1.0
        float x = hiprand_uniform(&state); // random x coordinate
        float y = hiprand_uniform(&state); // random y coordinate

        // check if the point is within the unit circle
        if (x * x + y * y <= 1.0f) {
            local_count++;
        }
    }

    // store local count in shared memory
    shared_counts[tid] = local_count;
    __syncthreads(); // synchronize threads within the block

    // perform parallel reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            shared_counts[tid] += shared_counts[tid + s];
        }
        __syncthreads(); // synchronize after each reduction step
    }

    // only the first thread in the block writes the total count to global memory
    if (tid == 0) {
        block_results[block_id] = shared_counts[0];
    }
}

int main(int argc, char** argv) {
    unsigned long long total_samples = 1ULL << 30; // default total samples (~1 billion)
    if (argc > 1) {
        try {
            total_samples = std::stoull(argv[1]);
        } catch (const std::invalid_argument& ia) {
            std::cerr << "Invalid argument: " << argv[1] << ". Please provide a number." << std::endl;
            return 1;
        }
    }

    // get device properties
    int device_id;
    CUDA_CHECK(hipGetDevice(&device_id));
    hipDeviceProp_t props; 
    CUDA_CHECK(hipGetDeviceProperties(&props, device_id));

    // determine number of blocks based on multiprocessor count
    int num_blocks = props.multiProcessorCount * 32; 

    // calculate total threads and samples per thread
    unsigned long long total_threads = (unsigned long long)num_blocks * THREADS_PER_BLOCK;
    // use ceiling division to ensure at least total_samples are generated
    unsigned long long samples_per_thread = (total_samples + total_threads - 1) / total_threads; 
    unsigned long long actual_total_samples = samples_per_thread * total_threads;

    std::cout << "GPU: " << props.name << std::endl;
    std::cout << "Threads per block: " << THREADS_PER_BLOCK << std::endl;
    std::cout << "Number of blocks: " << num_blocks << std::endl;
    std::cout << "Samples per thread: " << samples_per_thread << std::endl;
    std::cout << "Total samples requested: " << total_samples << std::endl;
    std::cout << "Total samples actual:    " << actual_total_samples << std::endl;

    // host and device memory allocation for block results
    std::vector<unsigned long long> h_block_results(num_blocks);
    unsigned long long* d_block_results;
    CUDA_CHECK(hipMalloc(&d_block_results, num_blocks * sizeof(unsigned long long)));

    // cuda events for timing
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    // seed for random number generation
    unsigned long long seed = time(NULL);

    // record start time, launch kernel, record stop time
    CUDA_CHECK(hipEventRecord(start));
    estimate_pi_kernel<<<num_blocks, THREADS_PER_BLOCK>>>(seed, samples_per_thread, d_block_results);
    CUDA_CHECK(hipGetLastError()); // check for kernel launch errors
    CUDA_CHECK(hipEventRecord(stop));

    // copy results from device to host
    CUDA_CHECK(hipMemcpy(h_block_results.data(), d_block_results, num_blocks * sizeof(unsigned long long), hipMemcpyDeviceToHost));

    // sum up all block results
    unsigned long long total_in_circle = 0;
    for (int i = 0; i < num_blocks; ++i) {
        total_in_circle += h_block_results[i];
    }

    // calculate elapsed time
    float milliseconds = 0;
    CUDA_CHECK(hipEventSynchronize(stop)); // wait for stop event to complete
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));

    // calculate pi estimate and error
    double pi_estimate = 4.0 * static_cast<double>(total_in_circle) / static_cast<double>(actual_total_samples);
    double actual_pi = std::acos(-1.0);

    // print results
    std::cout << std::fixed << std::setprecision(12);
    std::cout << "\nPoints in circle: " << total_in_circle << std::endl;
    std::cout << "Kernel execution time: " << milliseconds << " ms" << std::endl;
    std::cout << "Estimated Pi = " << pi_estimate << std::endl;
    std::cout << "Actual Pi    = " << actual_pi << std::endl;
    std::cout << "Error        = " << std::abs(pi_estimate - actual_pi) << std::endl; 

    // clean up cuda resources
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
    CUDA_CHECK(hipFree(d_block_results));

    return 0;
}